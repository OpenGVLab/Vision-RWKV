#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"
#define EPS (1e-6)

template <typename F>
__global__ void kernel_forward(const int B, const int T, const int C, const int H,
                               const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const float *__restrict__ _w, const F *__restrict__ _u,
                               F *__restrict__ const _y)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    _u += h*_N_;

    __shared__ float r[_N_], k[_N_], u[_N_], w[_N_];
    float state2[_N_] = {0};

    __syncthreads();
    u[i] = float(_u[i]);
    __syncthreads();

    for (int t = (b+1)*T*C + h*_N_ + i - C; t >= b*T*C + h*_N_ + i; t -= C)
    {
        __syncthreads();
        w[i] = exp(_w[t]);
        r[i] = float(_r[t]);
        k[i] = float(_k[t]);
        __syncthreads();

        const float v = float(_v[t]);
        float y = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j+=4)
        {
            const float4& r_ = (float4&)(r[j]);
            const float4& k_ = (float4&)(k[j]);
            const float4& w_ = (float4&)(w[j]);
            float4& s = (float4&)(state2[j]);
            float4 x;

            x.x = k_.x * v;
            x.y = k_.y * v;
            x.z = k_.z * v;
            x.w = k_.w * v;
            y += r_.x * (s.x);
            y += r_.y * (s.y);
            y += r_.z * (s.z);
            y += r_.w * (s.w);
            s.x = s.x * w_.x + x.x;
            s.y = s.y * w_.y + x.y;
            s.z = s.z * w_.z + x.z;
            s.w = s.w * w_.w + x.w;
        }
        _y[t] = F(y);
    }
    float state1[_N_] = {0};
    for (int t = b*T*C + h*_N_ + i; t < (b+1)*T*C + h*_N_ + i; t += C)
    {
        __syncthreads();
        w[i] = exp(_w[t]);
        r[i] = float(_r[t]);
        k[i] = float(_k[t]);
        __syncthreads();

        const float v = float(_v[t]);
        float y = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j+=4)
        {
            const float4& r_ = (float4&)(r[j]);
            const float4& k_ = (float4&)(k[j]);
            const float4& w_ = (float4&)(w[j]);
            const float4& u_ = (float4&)(u[j]);
            float4& s = (float4&)(state1[j]);
            float4 x;

            x.x = k_.x * v;
            x.y = k_.y * v;
            x.z = k_.z * v;
            x.w = k_.w * v;

            y += r_.x * (u_.x * x.x + s.x);
            y += r_.y * (u_.y * x.y + s.y);
            y += r_.z * (u_.z * x.z + s.z);
            y += r_.w * (u_.w * x.w + s.w);

            s.x = s.x * w_.x + x.x;
            s.y = s.y * w_.y + x.y;
            s.z = s.z * w_.z + x.z;
            s.w = s.w * w_.w + x.w;
        }
        _y[t] += F(y);
    }
}

template <typename F>
__global__ void kernel_backward_111(const int B, const int T, const int C, const int H,
    const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const float *__restrict__ _w, const F *__restrict__ _u, const F *__restrict__ const _gy,
    F *__restrict__ const _gr, F *__restrict__ const _gk, F *__restrict__ const _gv, F *__restrict__ const _gu)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    _u += h*_N_;

    __shared__ float u_[_N_];
    __shared__ float r[_N_], k[_N_], v[_N_], w_[_N_], gy[_N_];
    __syncthreads();
    u_[i] = float(_u[i]);
    __syncthreads();

    const float u = u_[i];

    float state1[_N_] = {0}, state2[_N_] = {0};
    float scccc1[_N_] = {0}, scccc2[_N_] = {0};
    float sdddd1[_N_] = {0}, sdddd2[_N_] = {0};

    const int t_0 = b*T*C + h*_N_ + i;
    const int t_T_1 = t_0 + (T-1)*C;
    const int t_T = t_0 + T*C;

    float gu = 0;
    for (int t = t_0; t < t_T; t += C)
    {
        __syncthreads();
        r[i] = float(_r[t]);
        v[i] = float(_v[t]);
        gy[i] = float(_gy[t]);
        k[i] = float(_k[t]);
        w_[i] = exp(_w[t]);
        __syncthreads();
        float gr = 0, gu_ = 0, gk = 0, gv = 0;;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s1 = state1[j];
            float& sc2 = scccc2[j];
            float& sd2 = sdddd2[j];
            float xk = r[i] * gy[j];
            float xv = r[j] * gy[i];
            float x = k[i] * v[j];

            gr += (u * x + s1) * gy[j];
            gu_ += x * gy[j];
            gk += sc2 * v[j];
            gv += sd2 * k[j];
            sc2 = sc2 * w_[i] + xk;
            sd2 = sd2 * w_[j] + xv;
            s1 = s1 * w_[i] + x;
        }
        _gr[t] = F(gr);
        _gk[t] = F(gk);
        _gv[t] = F(gv);
        gu += float(_r[t]) * gu_;
    }
    _gu[b*C + h*_N_ + i] = F(gu);


    for (int t = t_T_1; t >= t_0; t -= C)
    {
        __syncthreads();
        v[i] = float(_v[t]);
        gy[i] = float(_gy[t]);
        __syncthreads();

        const float rr = float(_r[t]);
        const float k = float(_k[t]);
        const float w = exp(_w[t]);
        float gk = 0, gr = 0;
        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& sc1 = scccc1[j];
            float& s2 = state2[j];
            float xk = rr * gy[j];
            float x2 = k * v[j];
            gr += gy[j] * s2;
            gk += (u * xk + sc1) * v[j];
            sc1 = xk + sc1 * w;
            s2 = x2 + s2 * w;
        }
        _gr[t] += F(gr);
        _gk[t] += F(gk);
    }

    for (int t = t_T_1; t >= t_0; t -= C)
    {
        __syncthreads();
        r[i] = float(_r[t]);
        k[i] = float(_k[t]);
        w_[i] = exp(_w[t]);
        __syncthreads();

        const float gyy = float(_gy[t]);
        float gv = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = sdddd1[j];
            float x = gyy * r[j];
            
            gv += (u_[j] * x + s) * k[j];
            s = x + s * w_[j];
        }
        _gv[t] += F(gv);
    }
}

template <typename F>
__global__ void kernel_backward_222(const int B, const int T, const int C, const int H,
    const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const float *__restrict__ _w, const F *__restrict__ _u, const F *__restrict__ const _gy,
    F *__restrict__ const _gw)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;

    __shared__ float v[_N_], gy[_N_];
    float saaaa[_N_] = {0}, sbbbb[_T_-2] = {0}, scccc[_N_] = {0};

    const int t_0 = b*T*C + h*_N_ + i;
    const int t_1 = t_0 + C;
    const int t_2 = t_0 + 2*C;
    const int t_T_1 = t_0 + (T-1)*C;

    for (int t = t_T_1; t > t_1; t -= C)
    {
        __syncthreads();
        gy[i] = float(_gy[t]);
        v[i] = float(_v[t-2*C]);
        __syncthreads();

        const float r = float(_r[t]);
        const float w = exp(_w[t-C]);
        float sum = 0.0f;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = saaaa[j];
            float x = r * gy[j];
            s = (s + x) * w;
            sum += s * v[j];
        }
        sbbbb[(t-t_2)/C] = sum * float(_k[t-2*C]);
    }

    float sss = sbbbb[0];
    _gw[t_0] = 0;
    _gw[t_1] = F(sss * _w[t_1]);

    for (int t = t_2; t < t_T_1; t += C)
    {
        __syncthreads();
        gy[i] = float(_gy[t]);
        v[i] = float(_v[t-2*C]);
        __syncthreads();

        const float w = exp(_w[t-C]);
        const float k = float(_k[t-2*C]);
        float sum = 0.0f;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = scccc[j];
            float x = k * v[j];
            s = (s + x) * w;
            sum += s * gy[j];
        }
        sss += sbbbb[(t-t_1)/C] - (sum * float(_r[t]));
        _gw[t] = F(sss * _w[t]);
    }
    _gw[t_T_1] = 0;
}

template <typename F>
__global__ void kernel_backward_333(const int B, const int T, const int C, const int H,
    const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const float *__restrict__ _w, const F *__restrict__ _u, const F *__restrict__ const _gy,
    F *__restrict__ const _gw)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;

    __shared__ float v[_N_], gy[_N_];
    float saaaa[_N_] = {0}, sbbbb[_T_-2] = {0}, scccc[_N_] = {0};

    const int t_0 = b*T*C + h*_N_ + i;
    const int t_T_1 = t_0 + (T-1)*C;
    const int t_T_2 = t_T_1 - C;
    const int t_T_3 = t_T_1 - 2*C;

    for (int t = t_0; t < t_T_2; t += C)
    {
        __syncthreads();
        gy[i] = float(_gy[t]);
        v[i] = float(_v[t+2*C]);
        __syncthreads();

        const float r = float(_r[t]);
        const float w = exp(_w[t+C]);
        float sum = 0.0f;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = saaaa[j];
            float x = r * gy[j];
            s = (s + x) * w;
            sum += s * v[j];
        }
        sbbbb[(t_T_3 - t)/C] = sum * float(_k[t+2*C]);
    }

    float sss = sbbbb[0];
    _gw[t_T_2] += F(sss * _w[t_T_2]);

    for (int t = t_T_3; t > t_0; t -= C)
    {
        __syncthreads();
        gy[i] = float(_gy[t]);
        v[i] = float(_v[t+2*C]);
        __syncthreads();

        const float w = exp(_w[t+C]);
        const float k = float(_k[t+2*C]);
        float sum = 0.0f;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = scccc[j];
            float x = k * v[j];
            s = (s + x) * w;
            sum += s * gy[j];
        }
        sss += sbbbb[(t_T_2-t)/C] - (sum * float(_r[t]));
        _gw[t] += F(sss * _w[t]);
    }
}

void cuda_forward(int B, int T, int C, int H, float *r, float *k, float *v, float *w, float *u, float *y)
{
    assert(H*_N_ == C);
    assert(_N_%4 == 0);
    kernel_forward<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, r, k, v, w, u, y);
}

void cuda_backward(int B, int T, int C, int H, float *r, float *k, float *v, float *w, float *u, float *gy, float *gr, float *gk, float *gv, float *gw, float *gu)
{
    assert(H*_N_ == C);
    assert(_N_%4 == 0);
    kernel_backward_111<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, r, k, v, w, u, gy, gr, gk, gv, gu);
    kernel_backward_222<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, r, k, v, w, u, gy, gw);
    kernel_backward_333<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, r, k, v, w, u, gy, gw);
}